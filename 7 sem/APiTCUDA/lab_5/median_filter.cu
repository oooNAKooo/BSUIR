#include "hip/hip_runtime.h"
#include "median_filter.h"
#include <opencv2/opencv.hpp>
#include <algorithm>
#include <cstdio>

// Загрузка изображения JPG
bool loadImageJPG(const char* filename, unsigned char** data, unsigned int* width, unsigned int* height) {
    cv::Mat image = cv::imread(filename, cv::IMREAD_GRAYSCALE);
    if (image.empty()) {
        fprintf(stderr, "Failed to load image: %s\n", filename);
        return false;
    }
    *width = image.cols;
    *height = image.rows;
    *data = new unsigned char[(*width) * (*height)];
    memcpy(*data, image.data, (*width) * (*height) * sizeof(unsigned char));
    printf("Image loaded: %s (%u x %u)\n", filename, *width, *height);
    return true;
}

// Сохранение изображения JPG
bool saveImageJPG(const char* filename, unsigned char* data, unsigned int width, unsigned int height) {
    cv::Mat image(height, width, CV_8UC1, data);
    if (!cv::imwrite(filename, image)) {
        fprintf(stderr, "Failed to save image: %s\n", filename);
        return false;
    }
    printf("Image saved: %s\n", filename);
    return true;
}

// Реализация медианного фильтра на CPU
void medianFilterCPU(const unsigned char* input, unsigned char* output, int width, int height) {
    for (int y = 1; y < height - 1; ++y) {
        for (int x = 1; x < width - 1; ++x) {
            unsigned char window[9];
            int idx = 0;
            for (int dy = -1; dy <= 1; ++dy) {
                for (int dx = -1; dx <= 1; ++dx) {
                    window[idx++] = input[(y + dy) * width + (x + dx)];
                }
            }
            std::sort(window, window + 9);
            output[y * width + x] = window[4];
        }
    }
}

// Проверка ошибок CUDA
void checkCudaErrors(hipError_t err) {
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

// Реализация медианного фильтра на GPU: без разделяемой памяти
__global__ void medianFilterKernel_NoShared(const unsigned char* input, unsigned char* output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= 1 && x < width - 1 && y >= 1 && y < height - 1) {
        unsigned char window[9];
        int idx = 0;
        for (int dy = -1; dy <= 1; ++dy) {
            for (int dx = -1; dx <= 1; ++dx) {
                window[idx++] = input[(y + dy) * width + (x + dx)];
            }
        }
        for (int i = 0; i < 5; ++i) {
            for (int j = i + 1; j < 9; ++j) {
                if (window[i] > window[j]) {
                    unsigned char temp = window[i];
                    window[i] = window[j];
                    window[j] = temp;
                }
            }
        }
        output[y * width + x] = window[4];
    }
}

void medianFilterGPU_NoShared(const unsigned char* input, unsigned char* output, int width, int height) {
    unsigned char *d_input, *d_output;

    size_t size = width * height * sizeof(unsigned char);
    checkCudaErrors(hipMalloc(&d_input, size));
    checkCudaErrors(hipMalloc(&d_output, size));
    checkCudaErrors(hipMemcpy(d_input, input, size, hipMemcpyHostToDevice));

    dim3 threads(16, 16);
    dim3 blocks((width + threads.x - 1) / threads.x, (height + threads.y - 1) / threads.y);

    medianFilterKernel_NoShared<<<blocks, threads>>>(d_input, d_output, width, height);
    checkCudaErrors(hipDeviceSynchronize());

    checkCudaErrors(hipMemcpy(output, d_output, size, hipMemcpyDeviceToHost));
    hipFree(d_input);
    hipFree(d_output);
}

// Реализация медианного фильтра на GPU: с разделяемой памятью
// Реализация аналогична, но с использованием __shared__ памяти.
