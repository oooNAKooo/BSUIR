#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h> //cuda api
#include <> // параметры ядра КУДА
#include <cstdlib> // рандом
#include <ctime> // генератор рандома
#include <chrono>  // Для замеров времени на CPU

#define BLOCK_SIZE 32 // размер блока (каждый блок содержит 32 потока, кот вып парал опер)

// Функция для проверки ошибок CUDA
void checkCudaError(hipError_t result, const char* msg) {
    if (result != hipSuccess) {
        std::cerr << "CUDA Error: " << msg << " - " << hipGetErrorString(result) << std::endl;
        exit(EXIT_FAILURE);
    }
}

// Инициализация матрицы случайными числами (0-99)
void initializeMatrix(int* matrix, int M, int N) {
    for (int i = 0; i < M * N; i++) {
        matrix[i] = rand() % 100;
    }
}

// CPU-реализация преобразования матрицы MxN с блоками 2x2 -> 1x4 (прямой порядок)
void transformCPU(const int* input, int* output, int M, int N) {
    int blockIdx = 0;  // Индекс блока 1x4 в выходной матрице
    for (int i = 0; i < M; i += 2) {
        for (int j = 0; j < N; j += 2) {
            // Копируем блок 2x2 в блок 1x4 (прямой порядок)
            output[blockIdx] = input[i * N + j];        // 1
            output[blockIdx + 1] = input[i * N + j + 1]; // 2
            output[blockIdx + 2] = input[(i + 1) * N + j]; // 3
            output[blockIdx + 3] = input[(i + 1) * N + j + 1]; // 4
            blockIdx += 4;  // Переходим к следующему блоку 1x4
        }
    }
}

// GPU-ядро для преобразования блоков 2x2 -> 1x4 (прямой порядок)
__global__ void transformGPU(const int* input, int* output, int M, int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int totalBlocks = (M / 2) * (N / 2);  // Общее количество блоков 1x4

    // Проверка выхода за пределы массива
    if (idx >= totalBlocks) return;

    int i = (idx / (N / 2)) * 2;  // Нахождение координат исходного блока в матрице
    int j = (idx % (N / 2)) * 2;

    // Преобразование 2x2 в 1x4
    output[idx * 4] = input[i * N + j];        // 1
    output[idx * 4 + 1] = input[i * N + j + 1]; // 2
    output[idx * 4 + 2] = input[(i + 1) * N + j]; // 3
    output[idx * 4 + 3] = input[(i + 1) * N + j + 1]; // 4
}

int main() {
    srand(static_cast<unsigned int>(time(0))); // Приведение к unsigned int

    // Размеры исходной матрицы
    const int M = 2048;  // Количество строк
    const int N = 4096;  // Количество столбцов

    // Размеры выходной матрицы
    const int outputM = M / 2;
    const int outputN = (N / 2) * 4;

    // Выделение памяти динамически на CPU
    int* inputCPU = new int[M * N];
    int* outputCPU = new int[outputM * outputN];
    int* outputGPUCPU = new int[outputM * outputN]; // Массив для результата GPU
    int* inputGPU, * outputGPU;

    // Инициализация входной матрицы
    initializeMatrix(inputCPU, M, N);

    // Проверка доступной памяти на GPU
    size_t freeMem = 0, totalMem = 0;
    hipMemGetInfo(&freeMem, &totalMem);
    std::cout << "Free GPU Memory: " << freeMem / (1024 * 1024) << " MB" << std::endl;
    std::cout << "Total GPU Memory: " << totalMem / (1024 * 1024) << " MB" << std::endl;

    // Замер времени выполнения на CPU
    auto startCPU = std::chrono::high_resolution_clock::now();
    transformCPU(inputCPU, outputCPU, M, N);
    auto endCPU = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> durationCPU = endCPU - startCPU;

    // Вывод времени выполнения CPU в миллисекундах
    std::cout << "Time CPU: " << durationCPU.count() << " ms" << std::endl;

    // Выделение памяти на GPU
    checkCudaError(hipMalloc(&inputGPU, M * N * sizeof(int)), "Allocating inputGPU");
    checkCudaError(hipMalloc(&outputGPU, outputM * outputN * sizeof(int)), "Allocating outputGPU");

    // Копирование данных на GPU (не учитываем в замере времени вычислений)
    checkCudaError(hipMemcpy(inputGPU, inputCPU, M * N * sizeof(int), hipMemcpyHostToDevice), "Copying to inputGPU");

    // Замер времени выполнения ядра на GPU
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    // Запуск ядра на GPU
    dim3 block(BLOCK_SIZE);
    dim3 grid((M * N / 4 + BLOCK_SIZE - 1) / BLOCK_SIZE);
    transformGPU << <grid, block >> > (inputGPU, outputGPU, M, N);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Измерение времени выполнения ядра (без учета копирования)
    float millisecondsGPU = 0;
    hipEventElapsedTime(&millisecondsGPU, start, stop);

    // Копирование результата обратно на CPU (после замера времени)
    checkCudaError(hipMemcpy(outputGPUCPU, outputGPU, outputM * outputN * sizeof(int), hipMemcpyDeviceToHost), "Copying from outputGPU");

    // Вывод времени выполнения GPU (без времени на передачу данных)
    std::cout << "Time GPU (without memory transfer): " << millisecondsGPU << " ms" << std::endl;

    // Сравнение результатов CPU и GPU
    bool isEqual = true;
    for (int i = 0; i < outputM * outputN; i++) {
        if (outputCPU[i] != outputGPUCPU[i]) {
            isEqual = false;
            break;
        }
    }

    if (isEqual) {
        std::cout << "Results match!" << std::endl;
    }
    else {
        std::cout << "Results do not match!" << std::endl;
    }

    // Освобождение памяти
    hipFree(inputGPU);
    hipFree(outputGPU);
    delete[] inputCPU;
    delete[] outputCPU;
    delete[] outputGPUCPU;

    return 0;
}
