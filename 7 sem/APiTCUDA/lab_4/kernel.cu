#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include <algorithm>
#include <chrono>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>

const int arraySize = 1000;  // Размер массива

// Функция для генерации случайных чисел
void generateRandomArray(float* array, int size) {
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<> dis(0.0, 1000.0);  // Диапазон чисел от 0 до 1000

    for (int i = 0; i < size; ++i) {
        array[i] = dis(gen);  // Генерация случайного числа
    }
}

// Сортировка на CPU
void sortArrayCPU(float* array, int size) {
    std::sort(array, array + size);  // Стандартная сортировка массива
}

// Функция для проверки результата сортировки
void printFirst10(float* array) {
    std::cout << "First 10 sorted numbers:" << std::endl;
    for (int i = 0; i < 10; ++i) {
        std::cout << array[i] << " ";
    }
    std::cout << std::endl;
}

// CUDA сортировка (на GPU без Thrust)
__global__ void sortArrayGPU(float* array, int size) {
    extern __shared__ float sharedArray[];

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < size) {
        sharedArray[threadIdx.x] = array[index];
        __syncthreads();

        for (int step = 1; step < blockDim.x; step *= 2) {
            if (threadIdx.x % (2 * step) == 0) {
                if (threadIdx.x + step < blockDim.x) {
                    if (sharedArray[threadIdx.x] > sharedArray[threadIdx.x + step]) {
                        float temp = sharedArray[threadIdx.x];
                        sharedArray[threadIdx.x] = sharedArray[threadIdx.x + step];
                        sharedArray[threadIdx.x + step] = temp;
                    }
                }
            }
            __syncthreads();
        }

        array[index] = sharedArray[threadIdx.x];
    }
}

int main() {
    // Создание массива на CPU
    float* h_array = new float[arraySize];
    generateRandomArray(h_array, arraySize);  // Генерация случайных чисел на CPU

    // Сортировка на CPU
    auto start = std::chrono::high_resolution_clock::now();
    sortArrayCPU(h_array, arraySize);
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> cpu_duration = end - start;  // Время в миллисекундах
    std::cout << "CPU sort time: " << cpu_duration.count() << " ms" << std::endl;

    // Печать первых 10 чисел после сортировки на CPU
    printFirst10(h_array);

    // Сортировка на GPU с использованием Thrust
    thrust::device_vector<float> d_array(arraySize);
    thrust::copy(h_array, h_array + arraySize, d_array.begin());  // Копирование данных в память GPU

    start = std::chrono::high_resolution_clock::now();
    thrust::sort(d_array.begin(), d_array.end());  // Сортировка на GPU с использованием Thrust
    end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> gpu_thrust_duration = end - start;  // Время в миллисекундах
    std::cout << "GPU Thrust sort time: " << gpu_thrust_duration.count() << " ms" << std::endl;

    // Копирование отсортированных данных обратно на CPU
    thrust::copy(d_array.begin(), d_array.end(), h_array);

    // Печать первых 10 чисел после сортировки на GPU (Thrust)
    printFirst10(h_array);

    // Сортировка на GPU без использования Thrust
    float* d_array_gpu;
    hipMalloc(&d_array_gpu, arraySize * sizeof(float));
    hipMemcpy(d_array_gpu, h_array, arraySize * sizeof(float), hipMemcpyHostToDevice);  // Копирование данных в память GPU

    start = std::chrono::high_resolution_clock::now();
    sortArrayGPU<<<(arraySize + 255) / 256, 256, 256 * sizeof(float)>>>(d_array_gpu, arraySize);  // Запуск сортировки на GPU
    hipDeviceSynchronize();  // Ожидание завершения вычислений
    end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> gpu_manual_duration = end - start;  // Время в миллисекундах
    std::cout << "GPU (manual) sort time: " << gpu_manual_duration.count() << " ms" << std::endl;

    // Копирование отсортированных данных обратно на CPU
    hipMemcpy(h_array, d_array_gpu, arraySize * sizeof(float), hipMemcpyDeviceToHost);

    // Печать первых 10 чисел после сортировки на GPU (без Thrust)
    printFirst10(h_array);

    // Освобождение памяти
    delete[] h_array;
    hipFree(d_array_gpu);

    return 0;
}
